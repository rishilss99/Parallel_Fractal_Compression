#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "parallel_compressor.h"

struct GlobalConstants {
    int imageWidth;
    int imageHeight;
    int rangeSize;
    int domainSize;
    int* imageData;
};

__constant__ GlobalConstants deviceConstants;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void resizeKernel(int* resizedImg, float scale, int w, int h) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % w;
  int y = index / w;

  if (x >= w || y >= h) {
    return;
  }

  int r, g, b, a;
  int oldX = scale * x;
  int oldY = scale * y;

  pixelGet(oldX, oldY, deviceConstants.imageWidth, deviceConstants.imageHeight, &r, &g, &b, &a, deviceConstants.imageData);
  pixelSet(x, y, w, h, r, g, b, a, resizedImg);
}

__global__ void transformKernel(int* fullImg, float scale, int widthInBlocks, int* codebookElements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = (index / (8 * widthInBlocks)) * deviceConstants.rangeSize;
  int domainIndex = (index / 8);
  int y = (domainIndex % widthInBlocks) * deviceConstants.rangeSize;
  int transform = index % 8;
  int* myElement = codebookElements + index * (4 * deviceConstants.rangeSize * deviceConstants.rangeSize);

  if (x >= (deviceConstants.imageWidth * scale) || y >= (deviceConstants.imageHeight * scale)) {
    return;
  }

  switch (transform) {
    case identity:
        identityTransform(x, y, fullImg, scale, myElement);
        break;
    case rot90:
        rot90Transform(x, y, fullImg, scale, myElement);
        break;
    case rot180:
        rot180Transform(x, y, fullImg, scale, myElement);
        break;
    case rot270:
        rot270Transform(x, y, fullImg, scale, myElement);
        break;
    case flip:
        flipTransform(x, y, fullImg, scale, myElement);
        break;
    case frot90:
        frot90Transform(x, y, fullImg, scale, myElement);
        break;
    case frot180:
        frot180Transform(x, y, fullImg, scale, myElement);
        break;
    case frot270:
        frot270Transform(x, y, fullImg, scale, myElement);
        break;
  }
}

__global__ void distanceKernel(int* codebookElements, int numCodebookElements, int widthInRangeBlocks,
                               int* distances, int* brightnesses, float* contrasts) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int rangeNum = index / numCodebookElements;
  int rSize = deviceConstants.rangeSize;
  int rangeX = (rangeNum % widthInRangeBlocks) * rSize;
  int rangeY = (rangeNum / widthInRangeBlocks) * rSize;
  int codebookNum = index % numCodebookElements;
  int* codebookPtr = codebookElements + codebookNum * (4 * deviceConstants.rangeSize * deviceConstants.rangeSize);

  if (rangeX >= deviceConstants.imageWidth || rangeY >= deviceConstants.imageHeight) {
    return;
  }

  // Get contrast scaling factor
  int domNorm = imDot(codebookPtr, 0, 0, rSize, rSize, codebookPtr, rSize, rSize, rSize, rSize, 0);
  float con;
  if (domNorm == 0) {
      con = 0;
  } else {
      float numerator = imDot(deviceConstants.imageData, rangeX, rangeY, deviceConstants.imageWidth,
                              deviceConstants.imageHeight, codebookPtr, rSize, rSize, rSize, rSize, 0);
      con = numerator / domNorm;
  }
  
  // Calculate brightness offset
  int rangeBrightness = averageBrightness(deviceConstants.imageData, rangeX, rangeY, deviceConstants.imageWidth,
                                          deviceConstants.imageHeight, rSize, rSize, 0);
  int codebookBrightness = averageBrightness(codebookPtr, 0, 0, rSize, rSize, rSize, rSize, 0);
  int bright = rangeBrightness - con * codebookBrightness;
  
  // Calculate distance
  int dist = 0;
  int colors[4];
  int otherColors[4];
  for (int y = 0; y < rSize; y++) {
      for (int x = 0; x < rSize; x++) {
          pixelGet(x + rangeX, y + rangeY, deviceConstants.imageWidth, deviceConstants.imageHeight,
                   colors, colors+1, colors+2, colors+3, deviceConstants.imageData);
          pixelGet(x, y, rSize, rSize, otherColors, otherColors+1, otherColors+2, otherColors+3, codebookPtr);
          otherColors[0] = con * otherColors[0] + bright;
          int diff = colors[0] - otherColors[0];
          dist += diff * diff;
      }
  }

  distances[index] = dist;
  brightnesses[index] = bright;
  contrasts[index] = con;
}

__global__ void bestMatchKernel(int* distances, int* brightnesses, float* contrasts, float scale, int numCodebookElements,
                                int numRangeBlocks, RangeBlockInfo* deviceRanges, CodebookElement* deviceBestCodebook) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= numRangeBlocks) {
    return;
  }

  int minDist = -1;
  int minElement = -1;
  for (int i = index * numCodebookElements; i < (index + 1) * numCodebookElements; ++i) {
    int curDist = distances[i];
    if (minElement < 0 || (curDist < minDist && curDist >= 0)) {
      minElement = i;
      minDist = curDist;
    }
  }

  deviceRanges[index].brightnessOffset = brightnesses[minElement];
  deviceRanges[index].contrastFactor = contrasts[minElement];
  int rangeWidth = deviceConstants.imageWidth / deviceConstants.rangeSize;
  deviceRanges[index].x = (index % rangeWidth) * deviceConstants.rangeSize;
  deviceRanges[index].y = (index / rangeWidth) * deviceConstants.rangeSize;

  int domainWidth = deviceConstants.imageWidth / deviceConstants.domainSize;
  int bestChoice = minElement - (index * numCodebookElements);
  int domainNum = bestChoice / 8;
  deviceBestCodebook[index].x = (domainNum / domainWidth) * deviceConstants.domainSize;
  deviceBestCodebook[index].y = (domainNum % domainWidth) * deviceConstants.domainSize;
  deviceBestCodebook[index].transform = (Transform) (bestChoice % 8);
}

CudaCompressor::CudaCompressor(const std::string& imageFilename, int rangeSize, int domainSize) {
  bestCodebook = NULL;
  image = readPPMImage(imageFilename.c_str());
  this->compIm.rangeSize = rangeSize;
  this->compIm.domainSize = domainSize;
  this->compIm.width = image->width;
  this->compIm.height = image->height;

  hipMalloc(&(cudaImageData), sizeof(int) * 4 * image->width * image->height);

  GlobalConstants hostConstants;
  hostConstants.imageWidth = image->width;
  hostConstants.imageHeight = image->height;
  hostConstants.rangeSize = rangeSize;
  hostConstants.domainSize = domainSize;
  hostConstants.imageData = cudaImageData;

  hipMemcpy(hostConstants.imageData, image->data, sizeof(int) * 4 * image->width * image->height, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(deviceConstants), &hostConstants, sizeof(GlobalConstants));
}

CudaCompressor::~CudaCompressor() {
  if (image) {
    delete image;
    hipFree(cudaImageData);
  }
  if (bestCodebook) {
    free(bestCodebook);
  }
}

void CudaCompressor::compress() {
  if (!image || image->width % compIm.rangeSize || image->height % compIm.rangeSize ||
      image->width % compIm.domainSize || image->height % compIm.domainSize) {
    std::cerr << "Invalid compression request" << std::endl;
    return;
  }
  dim3 rangeDim(compIm.rangeSize * compIm.rangeSize, 1);

  // Get resized image
  int* smallImg;
  float scale = ((float) compIm.domainSize) / compIm.rangeSize;
  int newW = image->width / scale;
  int newH = image->height / scale;
  
  hipMalloc(&(smallImg), sizeof(int) * 4 * newW * newH);

  dim3 resizeDim((newW * newH) / rangeDim.x);
  resizeKernel<<<resizeDim, rangeDim>>>(smallImg, scale, newW, newH);
  hipDeviceSynchronize();

  // Make Codebook elements
  int* codebookElements;
  int numDomainBlocks = (image->width / compIm.domainSize) * (image->height / compIm.domainSize);
  hipMalloc(&(codebookElements), sizeof(int) * 4 * compIm.rangeSize * compIm.rangeSize * numDomainBlocks * 8);
  dim3 baseDim(1024, 1);
  dim3 transformDim(((numDomainBlocks * 8) / baseDim.x) + 1);
  transformKernel<<<transformDim, baseDim>>>(smallImg, 1 / scale, image->width / compIm.domainSize, codebookElements);
  hipDeviceSynchronize();

  // Calculate range block-codebook element pairwise distances
  int* distances;
  int* brightnesses;
  float* contrasts;
  int numCodebookElements = numDomainBlocks * 8;
  int numRangeBlocks = (image->width / compIm.rangeSize) * (image->height / compIm.rangeSize);
  hipMalloc(&(distances), sizeof(int) * numCodebookElements * numRangeBlocks);
  hipMalloc(&(brightnesses), sizeof(int) * numCodebookElements * numRangeBlocks);
  hipMalloc(&(contrasts), sizeof(float) * numCodebookElements * numRangeBlocks);
  dim3 distDim(((numCodebookElements * numRangeBlocks) / baseDim.x) + 1);
  distanceKernel<<<distDim, baseDim>>>(codebookElements, numCodebookElements, image->width / compIm.rangeSize,
                                       distances, brightnesses, contrasts);
  hipDeviceSynchronize();

  // Fill the compressed image with the best pairs
  RangeBlockInfo* deviceRanges;
  CodebookElement* deviceBestCodebook;
  hipMalloc(&(deviceRanges), sizeof(RangeBlockInfo) * numRangeBlocks);
  hipMalloc(&(deviceBestCodebook), sizeof(CodebookElement) * numRangeBlocks);
  dim3 bestDim((numRangeBlocks / baseDim.x) + 1);
  bestMatchKernel<<<bestDim, baseDim>>>(distances, brightnesses, contrasts, scale, numCodebookElements,
                                        numRangeBlocks, deviceRanges, deviceBestCodebook);
  hipDeviceSynchronize();

  bestCodebook = (CodebookElement*) malloc(sizeof(CodebookElement) * numRangeBlocks);
  compIm.rangeInfo.resize(numRangeBlocks);
  hipMemcpy(compIm.rangeInfo.data(), deviceRanges, sizeof(RangeBlockInfo) * numRangeBlocks, hipMemcpyDeviceToHost);
  hipMemcpy(bestCodebook, deviceBestCodebook, sizeof(CodebookElement) * numRangeBlocks, hipMemcpyDeviceToHost);
  for (int i = 0; i < numRangeBlocks; ++i) {
    compIm.rangeInfo[i].codebookElement = &(bestCodebook[i]);
  }

  /*int* hostDistances = (int*) malloc(sizeof(int) * numCodebookElements * numRangeBlocks);
  hipMemcpy(hostDistances, brightnesses, sizeof(int) * numCodebookElements * numRangeBlocks, hipMemcpyDeviceToHost);
  for (int i = 0; i < numCodebookElements * numRangeBlocks; ++i) {
    std::cout << hostDistances[i] << std::endl;
  }
  free(hostDistances);*/

  hipFree(smallImg);
  hipFree(codebookElements);
  hipFree(distances);
  hipFree(brightnesses);
  hipFree(contrasts);
  hipFree(deviceRanges);
  hipFree(deviceBestCodebook);
}

void CudaCompressor::saveToFile(const std::string& filename) {
  writeFracFile(compIm, filename.c_str());
}

CompressedImage* CudaCompressor::getCompressedContents() {
    return &compIm;
}